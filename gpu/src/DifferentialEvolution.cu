#include "hip/hip_runtime.h"
#include "DifferentialEvolution.cuh"

#include <chrono>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <vector>

#define gpuErrorCheck(ans)                                                     \
  { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPU assert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    getchar();
    if (abort)
      exit(code);
  }
}

namespace deGPU {

DifferentialEvolution::DifferentialEvolution(int populationSize,
                                             int numberOfGeneration, int dim,
                                             double CR, double F, double minX,
                                             double maxX, uint blockDim) {
  if (populationSize < 4) {
    throw std::invalid_argument(
        "populationSize must be greater than 4 (i.e populationSize>=4)");
  }
  _populationSize = populationSize;
  _numberOfGeneration = numberOfGeneration;
  _dim = dim;
  _pCR = CR;
  _pF = F;
  _minX = minX;
  _maxX = maxX;
  _blockDim = blockDim;

  gpuErrorCheck(
      hipMalloc((void **)&zD, _dim * _populationSize * sizeof(double)));
  gpuErrorCheck(
      hipMalloc((void **)&rD, _dim * _populationSize * sizeof(double)));
  gpuErrorCheck(
      hipMalloc((void **)&trialD, _dim * _populationSize * sizeof(double)));
  gpuErrorCheck(hipMalloc((void **)&populationD,
                           _dim * _populationSize * sizeof(double)));
  gpuErrorCheck(hipMalloc((void **)&newPopulationD,
                           _dim * _populationSize * sizeof(double)));
  costH = new double[_populationSize];
  gpuErrorCheck(hipMalloc((void **)&costD, _populationSize * sizeof(double)));
  // create random states
  gpuErrorCheck(
      hipMalloc((void **)&globalRandStatesD,
                 _populationSize * sizeof(hiprandStatePhilox4_32_10_t)));
}

optimizeResult DifferentialEvolution::optimize() {

  static constexpr float warpSize = 32;
  int blockDim = static_cast<int>(
      std::ceil(static_cast<float>(_blockDim) / warpSize) * warpSize);

  int gridDim = std::ceil(static_cast<float>(_populationSize) /
                          static_cast<float>(blockDim));

  // initialize population and cost for each agent(we use
  // hiprandStatePhilox4_32_10_t for performance reason)
  gpu::init<<<gridDim, blockDim>>>(
      populationD, _populationSize, _dim, costD, clock(),
      (hiprandStatePhilox4_32_10_t *)globalRandStatesD, _minX, _maxX);

  gpuErrorCheck(hipPeekAtLastError());
  gpuErrorCheck(hipDeviceSynchronize());

  for (int i = 0; i < _numberOfGeneration; ++i) {
    gpu::SelectionAndCrossing<<<gridDim, blockDim>>>(
        _populationSize, _dim, populationD, newPopulationD, zD, rD, trialD, _pF,
        _pCR, costD, (hiprandStatePhilox4_32_10_t *)globalRandStatesD, _minX,
        _maxX);
    gpuErrorCheck(hipPeekAtLastError());
    // swap population(for free)
    double *tmp = populationD;
    populationD = newPopulationD;
    newPopulationD = tmp;
  }

  gpuErrorCheck(hipDeviceSynchronize());

  // copy the cost from device to host
  gpuErrorCheck(hipMemcpy(costH, costD, _populationSize * sizeof(double),
                           hipMemcpyDeviceToHost));

  // find the best solution
  auto [minIndex, minCost] = returnBestSolution();

  // copy the best solution from device to host
  std::vector<double> bestX(_dim);
  hipMemcpy(bestX.data(), populationD + minIndex * _dim, _dim * sizeof(double),
             hipMemcpyDeviceToHost);

  return {minCost, bestX};
}

std::pair<int, double> DifferentialEvolution::returnBestSolution() {
  double minCost = costH[0];
  int minIndex = 0;
  for (int i = 1; i < _populationSize; ++i) {
    if (costH[i] < minCost) {
      minCost = costH[i];
      minIndex = i;
    }
  }
  return {minIndex, minCost};
}

DifferentialEvolution::~DifferentialEvolution() {
  gpuErrorCheck(hipFree(zD));
  gpuErrorCheck(hipFree(rD));
  gpuErrorCheck(hipFree(trialD));
  gpuErrorCheck(hipFree(populationD));
  gpuErrorCheck(hipFree(newPopulationD));
  gpuErrorCheck(hipFree(costD));
  gpuErrorCheck(hipFree(globalRandStatesD));
  delete[] costH;
}

} // namespace deGPU
